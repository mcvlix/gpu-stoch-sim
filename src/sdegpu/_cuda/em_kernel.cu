#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename T>
__global__ void em_kernel(const T* __restrict__ ftk,
                          const T* __restrict__ gtk,
                          const T* __restrict__ sqrt_dt,
                          const T* __restrict__ eps,
                          const T   dt,
                          T* __restrict__ y,
                          const int64_t N)
{
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        // y[i] += ftk[i] * dt + gtk[i] * (sqrt_dt[i] * eps[i]);
        T inc = ftk[i] * dt + gtk[i] * (sqrt_dt[i] * eps[i]);
        y[i] += inc;
    }
}

torch::Tensor em_step_cuda(torch::Tensor y,
                           torch::Tensor ftk,
                           torch::Tensor gtk,
                           torch::Tensor dt,        // scalar or broadcasted
                           torch::Tensor sqrt_dt,   // broadcasted to y shape
                           torch::Tensor eps)
{
    TORCH_CHECK(y.is_cuda(), "y must be CUDA");
    auto N = y.numel();
    auto dtype = y.scalar_type();

    // ensure contiguous
    y = y.contiguous();
    ftk = ftk.contiguous();
    gtk = gtk.contiguous();
    sqrt_dt = sqrt_dt.contiguous();
    eps = eps.contiguous();

    const int threads = 256;
    const int blocks = (N + threads - 1) / threads;

    if (dtype == torch::kFloat32) {
        float dt_scalar = dt.item<float>();
        em_kernel<<<blocks, threads>>>(
            ftk.data_ptr<float>(),
            gtk.data_ptr<float>(),
            sqrt_dt.data_ptr<float>(),
            eps.data_ptr<float>(),
            dt_scalar,
            y.data_ptr<float>(),
            N
        );
    } else if (dtype == torch::kFloat64) {
        double dt_scalar = dt.item<double>();
        em_kernel<<<blocks, threads>>>(
            ftk.data_ptr<double>(),
            gtk.data_ptr<double>(),
            sqrt_dt.data_ptr<double>(),
            eps.data_ptr<double>(),
            dt_scalar,
            y.data_ptr<double>(),
            N
        );
    } else {
        TORCH_CHECK(false, "Unsupported dtype");
    }

    return y;
}
